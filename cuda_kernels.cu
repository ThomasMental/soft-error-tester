#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"
#include "cuda_kernels.cuh"
#include <thread> 
#include <chrono> 
#include <cstdio>

__global__ void simple_write_kernel(float* data, int n_elements, float value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_elements) {
        data[idx] = value;
    }
}

__global__ void simple_read_and_compare_kernel(float* data, int n_elements, float value, int* error_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_elements) {
        if (data[idx] != value) {
            atomicAdd(error_count, 1);
        }
    }
}

__global__ void initialize_to_one(float* data, size_t block_size, size_t n_blocks) {
    size_t block_idx = blockIdx.x;
    size_t idx = threadIdx.x;
    size_t offset = block_idx * (block_size / sizeof(float));

    if (block_idx % 2 == 1) {  // write 1 to block of odd index
        for (size_t i = idx; i < block_size / sizeof(float); i += blockDim.x) {
            data[offset + i] = 1.0f;
        }
    }
}

__global__ void rand_write_kernel(float* data, size_t block_size, size_t n_blocks, unsigned long long seed) {
    size_t block_idx = blockIdx.x;
    size_t idx = threadIdx.x;
    size_t offset = block_idx * (block_size / sizeof(float));

    if (block_idx % 2 == 0) {  // write random value to block of even index
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        for (size_t i = idx; i < block_size / sizeof(float); i += blockDim.x) {
            data[offset + i] = hiprand_uniform(&state);
        }
    }
}

__global__ void verify_one_blocks_kernel(float* data, size_t block_size, size_t n_blocks, int* error_count) {
    size_t block_idx = blockIdx.x;
    size_t idx = threadIdx.x;
    size_t offset = block_idx * (block_size / sizeof(float));

    if (block_idx % 2 == 1) {  // verify the value of blocks of odd index
        for (size_t i = idx; i < block_size / sizeof(float); i += blockDim.x) {
            if (data[offset + i] != 1.0f) {
                atomicAdd(error_count, 1);
            }
        }
    }
}

// Reverse bits
__device__ unsigned int reverseBits(unsigned int n) {
    unsigned int reversed = 0;
    for (int i = 0; i < 32; i++) {
        reversed <<= 1;               // Shift the bits of the result left.
        reversed |= (n & 1);          // Add the least significant bit of n to reversed.
        n >>= 1;                      // Shift the bits of n right.
    }
    return reversed;
}

__global__ void reverse_bits_kernel(float* data, int n_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_elements) {
        // change the type from float* to unsigned int
        unsigned int* bits = reinterpret_cast<unsigned int*>(&data[idx]);
        unsigned int reversedBits = reverseBits(*bits);
        *bits = reversedBits;
    }
}

__global__ void verify_reverse_bits_kernel(float* data, int n_elements, int* error_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_elements) {
        unsigned int* bits = reinterpret_cast<unsigned int*>(&data[idx]);
        unsigned int reversedBits = reverseBits(*bits);
        unsigned int revertedBits = reverseBits(reversedBits);
        if (revertedBits != *bits) {
            atomicAdd(error_count, 1);
        }
    }
}


// Main CUDA kernel used to run all the tests
void run_kernels(float* data, size_t block_size_bytes, size_t n_blocks, size_t grid_size, size_t block_size, int* host_errors, float* elapsed_time, int test_type) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate and initialize error count
    int* error_count;
    hipMalloc((void**)&error_count, sizeof(int));
    hipMemset(error_count, 0, sizeof(int));

    // Start timing
    hipEventRecord(start);

    if (test_type == 1) {
        // Test 1: write the assigned value to all the memory and check 
        size_t total_elements = block_size_bytes * n_blocks;
        size_t grid_size = total_elements / block_size;
        
        for (int j = 0; j < 1000; j++) {
            float test_value = j;
            simple_write_kernel<<<grid_size, block_size>>>(data, total_elements / sizeof(float), test_value);
            hipDeviceSynchronize();
            simple_read_and_compare_kernel<<<grid_size, block_size>>>(data, total_elements / sizeof(float), test_value, error_count);
            hipDeviceSynchronize();
        }
    } 
    else if (test_type == 2) {
        // Test 2：write and verify blocks with even and odd index
        initialize_to_one<<<grid_size, block_size>>>(data, block_size_bytes, n_blocks);
        hipDeviceSynchronize();
        for (int j = 0; j < 1000; j++) {
            rand_write_kernel<<<grid_size, block_size>>>(data, block_size_bytes, n_blocks, time(NULL) + j);
            hipDeviceSynchronize();
        }
        verify_one_blocks_kernel<<<grid_size, block_size>>>(data, block_size_bytes, n_blocks, error_count);
        hipDeviceSynchronize();
    }
    else if (test_type == 3) {
        // Test 3: write the value and wait for 10 minutes
        float test_value = 1.0f;
        size_t total_elements = block_size_bytes * n_blocks;
        size_t grid_size = total_elements / block_size;
        simple_write_kernel<<<grid_size, block_size>>>(data, total_elements / sizeof(float), test_value);
        hipDeviceSynchronize();
        
        printf("Waiting for 10 minutes...\n");
        std::this_thread::sleep_for(std::chrono::minutes(10)); 

        simple_read_and_compare_kernel<<<grid_size, block_size>>>(data, total_elements / sizeof(float), test_value, error_count);
        hipDeviceSynchronize();
    }
    else if(test_type == 4) {
        for (int j = 0; j < 1000; j++) {
            size_t total_elements = block_size_bytes * n_blocks;
            size_t grid_size = total_elements / block_size;
            reverse_bits_kernel<<<grid_size, block_size>>>(data, total_elements / sizeof(float));
            hipDeviceSynchronize();
            verify_reverse_bits_kernel<<<grid_size, block_size>>>(data, total_elements / sizeof(float), error_count);
            hipDeviceSynchronize();
        }
    }
    
    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get elapsed time
    hipEventElapsedTime(elapsed_time, start, stop);

    // Copy error count back to host
    hipMemcpy(host_errors, error_count, sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(error_count);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
